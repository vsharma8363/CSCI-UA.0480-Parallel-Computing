#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define RANGE 11.79


/*** TODO: insert the declaration of the kernel function below this line ***/
int numBlocks = 4;
int numThreads = 500;

__global__ void vecGPU(float *ad, float *bd, float *cd, int size, int elementsPerThread, int totalNumThreads) {
	// Calculate thread ID
	int threadID = threadIdx.x + blockDim.x * blockIdx.x;
	// Calculate upper and lower bound of calculation for this thread
	int lower = elementsPerThread * threadID;
	int upper = lower + elementsPerThread;
	if (threadID == (totalNumThreads) - 1)
		upper = size;
	// Run calculation procedure for each thread
	for(int i = lower; i < upper; i++) {
		if (i < size)
			cd[i] += ad[i] * bd[i];
	}
}
/**** end of the kernel declaration ***/


int main(int argc, char *argv[]){

	int n = 0; //number of elements in the arrays
	int i;  //loop index
	float *a, *b, *c; // The arrays that will be processed in the host.
	float *temp;  //array in host used in the sequential code.
	float *ad, *bd, *cd; //The arrays that will be processed in the device.
	clock_t start, end; // to meaure the time taken by a specific part of code
	
	if(argc != 2){
		printf("usage:  ./vectorprog n\n");
		printf("n = number of elements in each vector\n");
		exit(1);
		}
		
	n = atoi(argv[1]);
	printf("Each vector will have %d elements\n", n);
	
	
	//Allocating the arrays in the host
	
	if( !(a = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array a\n");
	   exit(1);
	}
	
	if( !(b = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array b\n");
	   exit(1);
	}
	
	if( !(c = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array c\n");
	   exit(1);
	}
	
	if( !(temp = (float *)malloc(n*sizeof(float))) )
	{
	   printf("Error allocating array temp\n");
	   exit(1);
	}
	
	//Fill out the arrays with random numbers between 0 and RANGE;
	srand((unsigned int)time(NULL));
	for (i = 0; i < n;  i++){
        a[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		b[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		c[i] = ((float)rand()/(float)(RAND_MAX)) * RANGE;
		temp[i] = c[i]; //temp is just another copy of C
	}
	
    //The sequential part
	start = clock();
	for(i = 0; i < n; i++) {
		temp[i] += a[i] * b[i];
	}
	end = clock();
	printf("Total time taken by the sequential part = %lf\n", (double)(end-start)/(double)CLOCKS_PER_SEC);

    /******************  The start GPU part: Do not modify anything in main() above this line  ************/
	//The GPU part
	start = clock();
	
	//	1. allocate ad, bd, and cd in the device
	int size = n*sizeof(float);
	hipMalloc((void**)&ad, size);
	hipMalloc((void**)&bd, size);
	hipMalloc((void**)&cd, size);
	// 2. send a, b, and c to the device
	hipMemcpy(ad, a, size, hipMemcpyHostToDevice);
	hipMemcpy(bd, b, size, hipMemcpyHostToDevice);
	hipMemcpy(cd, c, size, hipMemcpyHostToDevice);
	// 4. Call the kernel (the kernel itself will be written at the comment at the end of this file), 
	//	   you need to write the number of threads, blocks, etc and their geometry.
	int elementsPerThread = (n / (numBlocks * numThreads));

	vecGPU<<<numBlocks, numThreads>>>(ad, bd, cd, n, elementsPerThread, numBlocks*numThreads);
	
	// 5. Bring the cd array back from the device and store it in c array (declared earlier in main)
	hipMemcpy(c, cd, size, hipMemcpyDeviceToHost);
	// 6. free ad, bd, and cd
	hipFree(ad); 
	hipFree(bd);
	hipFree(cd); 
	
	end = clock();
	printf("Total time taken by the GPU part = %lf\n", (double)(end-start)/ CLOCKS_PER_SEC);
	/******************  The end of the GPU part: Do not modify anything in main() below this line  ************/
	
	//checking the correctness of the GPU part
	for(i = 0; i < n; i++)
		if( fabsf(temp[i] - c[i]) >= 0.009) //compare up to the second degit in floating point
			printf("Element %d in the result array does not match the sequential version\n", i);
		
	// Free the arrays in the host
	free(a); free(b); free(c); free(temp);

	return 0;
}

/**** TODO: Write the kernel itself below this line *****/
